#include "hip/hip_runtime.h"
#include "bmm.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

// Keep your original tile sizes that showed better performance
#define TILEX 32
#define TILEY 8
#define TILEZ 32

__global__ void kernelFunc(float* ad, float* bd, float* cd, const int m, const int n) {
    // Shared memory
    __shared__ float AS[TILEY][TILEZ];
    __shared__ float BS[TILEZ][TILEX];
    
    // Calculate global indices
    const int i = TILEY * by + ty;
    const int j = TILEX * bx + tx;
    
    float temp_sum = 0.0f;
    
    // Precompute loop bound
    const int num_tiles = n / TILEZ;
    
    for (int p = 0; p < num_tiles; p++) {
        // Optimized loading of AS
        if (TILEX > TILEZ) {
            if (tx < TILEZ) {
                AS[ty][tx] = ad[i * n + (p * TILEZ + tx)];
            }
        } else {
            #pragma unroll
            for (int k = 0; k < TILEZ/TILEX; k++) {
                AS[ty][k * TILEX + tx] = ad[i * n + (p * TILEZ + k * TILEX + tx)];
            }
        }
        
        // Optimized loading of BS
        if (TILEZ > TILEY) {
            #pragma unroll
            for (int k = 0; k < TILEZ/TILEY; k++) {
                BS[k * TILEY + ty][tx] = bd[(p * TILEZ + k * TILEY + ty) * n + j];
            }
        } else {
            if (ty < TILEZ) {
                BS[ty][tx] = bd[(p * TILEZ + ty) * n + j];
            }
        }
        
        __syncthreads();
        
        // Accumulate product with loop unrolling
        #pragma unroll
        for (int q = 0; q < TILEZ; q++) {
            temp_sum += AS[ty][q] * BS[q][tx];
        }
        
        __syncthreads();
    }
    
    // Only write if within bounds
    if (i < n && j < n) {
        cd[i * n + j] = temp_sum;
    }
}

dim3 getDimGrid(const int m, const int n) {
    dim3 dimGrid((n + TILEX - 1) / TILEX, (n + TILEY - 1) / TILEY);
    return dimGrid;
}

dim3 getDimBlock(const int m, const int n) {
    dim3 dimBlock(TILEX, TILEY);
    return dimBlock;
}